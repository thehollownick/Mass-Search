#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <windows.h>
#include <iostream>
#include <fstream>
#include <sstream>
#include <time.h>
#include <ctime>

//calculating working hours
double PCFreq = 0.0;
__int64 CounterStart = 0;

void StartCounter()
{
	LARGE_INTEGER li;
	if (!QueryPerformanceFrequency(&li))
		std::cout << "QueryPerformanceFrequency failed!\n";

	PCFreq = double(li.QuadPart) / 1000.0;

	QueryPerformanceCounter(&li);
	CounterStart = li.QuadPart;
}
double GetCounter()
{
	LARGE_INTEGER li;
	QueryPerformanceCounter(&li);
	return double(li.QuadPart - CounterStart) / PCFreq;
}

__global__ void parallelGrep(char* global_data, int globalData_Size, char* key, int key_size, int* key_indexes, int* curr_index)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	int key0 = key[0];

	if (idx < globalData_Size)
	{
		if (global_data[idx] == key0)
		{
			int save = 1;

			for (int i = 1; i < key_size; i++)
			{
				if (global_data[idx + i] != key[i])
				{
					save = 0;
					break;
				}
			}
			if (save == 1)
			{
				key_indexes[atomicAdd(curr_index, 1)] = idx;
			}
		}
	}

}

int SubStrCount(const char* str, const char* subStr)
{
	int res = 0, len;
	if (str && subStr)
	{
		if (len = strlen(subStr))
		{
			for (char const* p = str; *p; p += strncmp(p, subStr, len) ? 1 : (++res, len))
				;
		}
	}
	return res;
}

int main()
{
	FILE* file;
	size_t numB;
	long lSize;
	size_t key_size, key_ind_size;
	char* data_d, *data_h, *key_d;
	char key_h[50];
	int key_ind_h[600];
	int* key_ind_d;
	int currind;
	int* curr_ind_h = &currind;
	int* curr_ind_d;


	file = fopen("text.txt", "rb");
	if (file == NULL)
	{
		printf("Cannot open txt file!\n");
		exit(1);
	}
	fseek(file, 0, SEEK_END);
	lSize = ftell(file);
	rewind(file);

	data_h = new char[lSize];	//  memory on host
	hipMalloc((void**)&data_d, lSize);	//  memory on device	



	numB = fread(data_h, 1, lSize, file); //read from file
	hipMemcpy(data_d, data_h, lSize, hipMemcpyHostToDevice);	// copy data to device memory

	strcpy(key_h, "can"); //substring
	key_size = strlen(key_h);

	hipMalloc((void**)&key_d, key_size);
	hipMemcpy(key_d, key_h, key_size, hipMemcpyHostToDevice); // copy substring to device memory

	//buff
	key_ind_size = sizeof(key_ind_h);
	memset(key_ind_h, 0, key_ind_size);
	hipMalloc((void**)&key_ind_d, key_ind_size);
	hipMemcpy(key_ind_d, key_ind_h, key_ind_size, hipMemcpyHostToDevice);

	*curr_ind_h = 0;
	hipMalloc((void**)&curr_ind_d, 4);
	hipMemcpy(curr_ind_d, curr_ind_h, 4, hipMemcpyHostToDevice);


	int block_size = 1024;
	int n_blocks = lSize / block_size + (lSize % block_size == 0 ? 0 : 1);

	///////////////////////////////////
	StartCounter();

	parallelGrep << < n_blocks, block_size >> > (data_d, lSize, key_d, key_size, key_ind_d, curr_ind_d);


	hipMemcpy(key_ind_h, key_ind_d, key_ind_size, hipMemcpyDeviceToHost);

	printf("Time GPU ");
	std::cout << GetCounter() << " ms" << std::endl;


	int pIter = 0;
	while (key_ind_h[pIter] != 0)
	{
		pIter++;
	}
	printf("%d", pIter);
	printf("%c", '\n');
	////host
	clock_t start = clock();
	int count = SubStrCount(data_h, key_h);
	printf("Time CPU = %lf ms\n", (long double)(clock() - start) / CLOCKS_PER_SEC * 1000);

	printf("%d", count);


	delete[] data_h;
	hipFree(data_d);
	hipFree(key_d);
	hipFree(key_ind_d);
	hipFree(curr_ind_d);
	return 0;
}
